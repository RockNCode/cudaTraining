#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>


__global__ void printThreadIds()
{
    printf("ThreadId.x = %d , ThreadId.y = %d , ThreadId.z = %d \n",
        threadIdx.x, threadIdx.y, threadIdx.z);
}

int main()
{
    int nx, ny;
    nx = 16;
    ny = 16;

    dim3 block(8, 8);
    dim3 grid(nx / block.x, ny / block.y);

    printThreadIds << <grid, block >> > ();
    hipDeviceSynchronize();
    hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}
